#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <random>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

using namespace std::chrono;

struct Square
{
    __host__ __device__
    float operator()(const float& x) const
    {
        return x * x;
    }
};

void reportTime(const char* msg, steady_clock::duration span)
{
    auto ms = duration_cast<milliseconds>(span);
    std::cout << msg << " - levou - " << ms.count() << " milisegundos" << std::endl;
}

float magnitude(const thrust::device_vector<float>& v)
{
    float init = 0.0f;
    Square unary_op;
    thrust::plus<float> binary_op;

    return std::sqrt(thrust::transform_reduce(v.begin(), v.end(), unary_op, init, binary_op));
}

int main(int argc, char** argv)
{
    if (argc != 2)
    {
        std::cerr << argv[0] << ": numero invalido de argumentos\n";
        std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n";
        return 1;
    }
    int n = std::atoi(argv[1]); //numero de elementos
    steady_clock::time_point ts, te;

    // Criando um vector em thrust
    thrust::device_vector<float> v_d(n);

    // Inicializando o vector
    ts = steady_clock::now();
    std::generate(v_d.begin(), v_d.end(), std::rand);
    te = steady_clock::now();
    reportTime("Inicializacao", te - ts);

    // Calculando a magnitude do vetor
    ts = steady_clock::now();
    float len = magnitude(v_d);
    te = steady_clock::now();
    reportTime("Tempo para calculo", te - ts);

    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << len << std::endl;

    return 0;
}
